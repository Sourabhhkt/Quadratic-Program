#include "hip/hip_runtime.h"
#include <stdio.h>
#include "support.h"
#include "kernel.cu"
#include <float.h>

int main(int argc, char**argv) {

    Timer timer;
    hipError_t cuda_ret;
    // Setting up input parameter for QP
    // W = np.array([
    //     [6, 3, 5, 0],
    //     [3, 6, 0, 1],
    //     [5, 0, 8, 0],
    //     [0, 1, 0, 10] 
    // ])
    // A = np.array([
    //     [3, -3, -2, 1],
    //     [4, 1, -1, -2] 
    // ])
    // E = np.array([
    //     [-1, 1, 0, 0],
    //     [3, 0, 1, 0] 
    // ])
    // C = np.array([-11,0,0,-5])
    // b = np.array([0,0])
    // l = np.array([-np.inf,-2])
    // h = np.array([-1,4])
    
    // W_inv = np.linalg.inv(W)

    //     ME_T = 
    //  [[-0.01105294  0.07155323]
    //  [ 0.03548575 -0.01919721]
    //  [-0.05759162  0.16230366]
    //  [ 0.02443281  0.05235602]]
    // s = 
    //  [0.26236184 0.26294357 0.2617801  0.52530541]

    // Initialize host variables ----------------------------------------------

    printf("\nSetting up the problem..."); fflush(stdout);
    startTime(&timer);
    
    int row_num = 4;
    int col_num = 2;

    // Constant parameter from the problem: l and h
    float* l; float* h; 
    float raw_l = {FLT_MIN, -2};float raw_h = {-1, 4};
    l = raw_l; h = raw_h;

    // Constant parameter from the problem: ME^T
    float** E = (float**) malloc( sizeof(float*)*col_num );
    for (int i = 0; i < row_num; i++){
        *(E+i) = (float*)malloc(sizeof(float)*row_num);
    }

    E[0][0] =  -1; E[0][1] =  1; E[0][2] =  0; E[0][3] =  0;
    E[1][0] =  3; E[1][1] =  0; E[1][2] =  1; E[1][3] =  0;
    printf("Matrix E: \n"); fflush(stdout);
    print_2d_array(row_num,col_num,E);

    // Constant parameter from the problem: ME^T
    float** ME_T = (float**) malloc( sizeof(float*)*row_num );
    for (int i = 0; i < row_num; i++){
        *(ME_T+i) = (float*)malloc(sizeof(float)*col_num);
    }

    ME_T[0][0] =  -0.01105294; ME_T[0][1] =  0.07155323;
    ME_T[1][0] =  0.03548575; ME_T[1][1] =  -0.01919721; 
    ME_T[2][0] =  -0.05759162; ME_T[2][1] =  0.16230366; 
    ME_T[3][0] =  0.02443281; ME_T[3][1] =  0.05235602; 
    printf("Matrix ME^T: \n"); fflush(stdout);
    print_2d_array(row_num,col_num,ME_T);

    // Constant parameter from the problem: s
    float raw_vecS[] = {0.26236184, 0.26294357, 0.2617801, 0.52530541};
    float* s = raw_vecS;
    
    printf("Vector s: \n"); fflush(stdout);
    print_1d_array(row_num,s);

    // Initialize u and x variable
    float raw_u[] = {10, -10};float raw_u_minus[] = {-10, 10};
    float* u_0 = raw_u;
    float* u_0_minus = raw_u_minus;

    float* x = (float*)malloc(sizeof(float)*row_num);
    x = vec_add_vec(row_num,mat_mul_vec(row_num, col_num, ME_T, u_0),s);
    printf("Vector ME^T*u0 + s: \n"); fflush(stdout);
    print_1d_array(row_num,x);

    // Initialize Ex
    float* Ex = (float*)malloc(sizeof(float)*col_num);
    Ex = mat_mul_vec(col_num,row_num, E, x);
    printf("Vector Ex: \n"); fflush(stdout);
    print_1d_array(col_num,Ex);

    // Initialize g_Ex_u
    float* g_Ex_u = (float*)malloc(sizeof(float)*row_num);
    g_Ex_u = g_function(row_num, vec_add_vec(row_num,Ex,u_0_minus))
    printf("Vector g_Ex_u: \n"); fflush(stdout);
    print_1d_array(col_num,g_Ex_u);
    // bool tolerance_met = false;
    // while (!tolerance_met)
    // {

    // }

    // Allocate device variables ----------------------------------------------

    // printf("Allocating device variables..."); fflush(stdout);
    // startTime(&timer);

    // float* u_d;
    // cuda_ret = hipMalloc((void**) &u_d, sizeof(float)*col_num);
	// if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");

    // float* u_prev_d;
    // cuda_ret = hipMalloc((void**) &u_prev_d, sizeof(float)*col_num);
	// if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");




    // //INSERT CODE HERE for B and C
    // float* B_d;
    // cuda_ret = hipMalloc((void**) &B_d, sizeof(float)*n);
	// if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");

    // float* C_d;
    // cuda_ret = hipMalloc((void**) &C_d, sizeof(float)*n);
	// if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");

    // hipDeviceSynchronize();




    // float* A_h = (float*) malloc( sizeof(float)*n );
    // for (unsigned int i=0; i < n; i++) { A_h[i] = (rand()%100)/100.00; }

    // float* B_h = (float*) malloc( sizeof(float)*n );
    // for (unsigned int i=0; i < n; i++) { B_h[i] = (rand()%100)/100.00; }

    // float* C_h = (float*) malloc( sizeof(float)*n );

    // stopTime(&timer); printf("%f s\n", elapsedTime(timer));
    // printf("    Vector size = %u\n", n);

    // // Allocate device variables ----------------------------------------------

    // printf("Allocating device variables..."); fflush(stdout);
    // startTime(&timer);

    // float* A_d;
    // cuda_ret = hipMalloc((void**) &A_d, sizeof(float)*n);
	// if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");

    // //INSERT CODE HERE for B and C
    // float* B_d;
    // cuda_ret = hipMalloc((void**) &B_d, sizeof(float)*n);
	// if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");

    // float* C_d;
    // cuda_ret = hipMalloc((void**) &C_d, sizeof(float)*n);
	// if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");

    // hipDeviceSynchronize();
    // stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // // Copy host variables to device ------------------------------------------

    // printf("Copying data from host to device..."); fflush(stdout);
    // startTime(&timer);

    // cuda_ret = hipMemcpy(A_d, A_h, sizeof(float)*n, hipMemcpyHostToDevice);
	// if(cuda_ret != hipSuccess) FATAL("Unable to copy memory to device");

    // //INSERT CODE HERE for B
    // cuda_ret = hipMemcpy(B_d, B_h, sizeof(float)*n, hipMemcpyHostToDevice);
	// if(cuda_ret != hipSuccess) FATAL("Unable to copy memory to device");

    // hipDeviceSynchronize();
    // stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // // Launch kernel ----------------------------------------------------------

    // printf("Launching kernel..."); fflush(stdout);
    // startTime(&timer);

    // const unsigned int THREADS_PER_BLOCK = 512;
    // const unsigned int numBlocks = (n - 1)/THREADS_PER_BLOCK + 1;
    // dim3 gridDim(numBlocks, 1, 1), blockDim(THREADS_PER_BLOCK, 1, 1);
    // //INSERT CODE HERE to call kernel
    // vecAddKernel<<<ceil(numBlocks),THREADS_PER_BLOCK>>>(A_d, B_d, C_d, n);

    // cuda_ret = hipDeviceSynchronize();
	// if(cuda_ret != hipSuccess) FATAL("Unable to launch kernel");
    // stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // // Copy device variables from host ----------------------------------------

    // printf("Copying data from device to host..."); fflush(stdout);
    // startTime(&timer);

    // //INSERT CODE HERE to copy C
    // cuda_ret = hipMemcpy(C_h, C_d, sizeof(float)*n, hipMemcpyDeviceToHost);
	// if(cuda_ret != hipSuccess) FATAL("Unable to copy memory to device");

    // hipDeviceSynchronize();
    // stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // // Verify correctness -----------------------------------------------------

    // printf("Verifying results..."); fflush(stdout);

    // verify(A_h, B_h, C_h, n);

    // // Free memory ------------------------------------------------------------

    // free(A_h);
    // free(B_h);
    // free(C_h);

    // //INSERT CODE HERE to free device matrices
    // hipFree(A_d);
    // hipFree(B_d);
    // hipFree(C_d);

    return 0;

}

